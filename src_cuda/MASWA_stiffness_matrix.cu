#include "hip/hip_runtime.h"
#include "MASW.cuh"

/* The function MASWaves_Ke_layer computes the element stiffness matrix
 of the j-th layer (j = 1,...,n) of the stratified earth
 model that is used in the inversion analysis. The stiffness matrix as a
 4x4 stored in a 1D Array. */

#define add(x,y) (hipCadd(x,y))
#define subtract(x,y) (hipCsub(x,y))
#define multiply(x,y) (hipCmul(x,y))
#define divide(x,y) (hipCdiv(x,y))

#define matrices(i,j,k) (matrices[i][j*size + k])

/*
*/
void MASWA_stiffness_matrix_CUDA(curve_t *curve, hipDoubleComplex **d_matrices){

    int size = 2*(curve->n+1);
    dfloat *d_c_test, *d_lambda, *d_h, *d_alpha, *d_beta, *d_rho;

    hipMalloc(&d_c_test, curve->velocities_length*sizeof(dfloat));
    hipMalloc(&d_lambda, curve->curve_length*sizeof(dfloat));
    hipMalloc(&d_h, curve->n*sizeof(dfloat));
    hipMalloc(&d_alpha, (curve->n+1)*sizeof(dfloat));
    hipMalloc(&d_beta, (curve->n+1)*sizeof(dfloat));
    hipMalloc(&d_rho, (curve->n+1)*sizeof(dfloat));

    hipMemcpy(d_c_test, curve->c_test, curve->velocities_length*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_lambda, curve->lambda_curve0, curve->curve_length*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_h, curve->h, curve->n*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, curve->alpha, (curve->n+1)*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, curve->beta, (curve->n+1)*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_rho, curve->rho, (curve->n+1)*sizeof(dfloat), hipMemcpyHostToDevice);

    // If the shared memory requirements for kernel_generate_stiffness_matrices are too large, then this can be reduced to 128:
    int blockSize = 256;
    int blocks = (curve->curve_length*curve->velocities_length / blockSize)+1;
    kernel_too_close<<<1,blockSize>>>(curve->velocities_length, curve->n+1, d_c_test, d_alpha, d_beta, 0.0001);

    // Form the stiffness matrices here:
    kernel_generate_stiffness_matrices<<<blocks, blockSize, 6*blockSize*sizeof(hipDoubleComplex)>>>(d_c_test, d_lambda, d_h, d_alpha, d_beta, d_rho, curve->n, curve->velocities_length, curve->curve_length, d_matrices);
    
    // Gaussian Elimination here:
    kernel_hepta_determinant_CUDA<<<blocks, blockSize, 4*blockSize*sizeof(hipDoubleComplex)>>>(curve->curve_length, curve->velocities_length, size, d_matrices);

    hipFree(d_c_test);
    hipFree(d_lambda);
    hipFree(d_h);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_rho);

}

/* Fills in a simple identity matrix for some test cases. Not used as part of MASW.
*/
__global__ void kernel_matrix_fill_in_serial(int velocities_length, int curve_length, int n, hipDoubleComplex **matrices){

    int size = 2*(n+1);
    

    for (int i=0; i<curve_length*velocities_length; ++i){
        for (int j=0; j<size; ++j){
            matrices(i,j,j) = make_hipDoubleComplex(1, 0);
        }
    }
}

/* Assigns matrix pointers to the contiguous chunk of memory where they are stored. This is necessary for the implementation
    utilizing cuBLAS (there may be a method that does not require a kernel, but other things I tried generated a seg fault).
    Since cuBLAS is no longer used, the code may be rewritten to omit this. (Matrices can just be stored as a 1D array without
    pointers)
*/
__global__ void kernel_assign_matrices_to_data(hipDoubleComplex **matrices, hipDoubleComplex *data, int curve_length, int velocities_length, int n){

    for (int i=0; i<curve_length*velocities_length; ++i){

        matrices[i] = (hipDoubleComplex*) ((char*) data+i*((size_t)(4*n*n + 8*n + 4))*sizeof(hipDoubleComplex));
    }
}

/* Performs Gaussian elimination on the stiffness matrices by taking advantage of their
    banded structure, unlike in the cuBLAS function. This version puts the current row
    used for elimination into shared memory, improving the speed of accessing it.
*/
__global__ void kernel_hepta_determinant_CUDA(int curve_length, int velocities_length, int size, hipDoubleComplex **matrices){

    int blockSize = blockDim.x;
    int threadIndex = threadIdx.x;
    int index = blockSize * blockIdx.x + threadIndex;
    int stride = blockSize * gridDim.x;

    int sharedIndex = 4*threadIndex;

    extern __shared__ hipDoubleComplex row[];

    for (int x=index; x<curve_length*velocities_length; x+=stride){

        for (int i=0; i<size; ++i){

            int end = i + 4;
            if (end > size){
                end = size;
            }

            for (int k=i; k<end; ++k){
                row[sharedIndex + k - i] = matrices(x,i,k);
            }

            // TODO: row switching

            //Gaussian elimination for the three rows (or fewer) below this one:
            for (int j=i+1; j<end; ++j){

                hipDoubleComplex coeff = hipCdiv(matrices(x,j,i), row[sharedIndex]);

                for (int k=i+1; k<end; ++k){

                    matrices(x,j,k) = hipCsub(matrices(x,j,k), hipCmul(coeff, row[sharedIndex + k - i]));
                }
            }
        }
    }

}








