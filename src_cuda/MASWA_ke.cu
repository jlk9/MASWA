#include "hip/hip_runtime.h"
#include "MASW.cuh"

/* The function MASWaves_Ke_layer computes the element stiffness matrix
 of the j-th layer (j = 1,...,n) of the stratified earth
 model that is used in the inversion analysis. The stiffness matrix as a
 4x4 stored in a 1D Array. */

/*
    Fills in entries of stiffness matrices utilizing multiple blocks, and takes advantage of the symmetry in Ke_layer and Ke_halfspace
*/
__global__ void kernel_generate_stiffness_matrices(dfloat *c_test, dfloat *lambda, dfloat *h, dfloat *alpha, dfloat *beta, dfloat *rho, int n, int velocities_length, int curve_length, hipDoubleComplex **matrices){

    int blockSize = blockDim.x;
    int threadIndex = threadIdx.x;
    int index = blockSize * blockIdx.x + threadIndex;
    int stride = blockSize * gridDim.x;
    int indexKe = threadIndex*6;

    extern __shared__ hipDoubleComplex Ke[];

    #define matrices(i,j,k) (matrices[i][j*2*(n+1) + k])

    for (int i=index; i<curve_length*velocities_length; i+=stride){

        dfloat k = 2*M_PI / lambda[i / velocities_length];

        for (int j=0; j<n; ++j){

            kernel_Ke_layer_symm(h[j], alpha[j], beta[j], rho[j], c_test[i % velocities_length], k, Ke+indexKe);
            
            matrices(i,(2*j),(2*j))     = hipCadd(matrices(i,(2*j),(2*j)), Ke[indexKe]);
            matrices(i,(2*j),(2*j+1))   = hipCadd(matrices(i,(2*j),(2*j+1)), Ke[indexKe + 1]);
            matrices(i,(2*j),(2*j+2))   = hipCadd(matrices(i,(2*j),(2*j+2)), Ke[indexKe + 2]);
            matrices(i,(2*j),(2*j+3))   = hipCadd(matrices(i,(2*j),(2*j+3)), Ke[indexKe + 3]);
            matrices(i,(2*j+1),(2*j+1)) = hipCadd(matrices(i,(2*j+1),(2*j+1)), Ke[indexKe + 4]);
            matrices(i,(2*j+1),(2*j+2)) = hipCsub(matrices(i,(2*j+1),(2*j+2)), Ke[indexKe + 3]);
            matrices(i,(2*j+1),(2*j+3)) = hipCadd(matrices(i,(2*j+1),(2*j+3)), Ke[indexKe + 5]);
            matrices(i,(2*j+2),(2*j+2)) = hipCadd(matrices(i,(2*j+2),(2*j+2)), Ke[indexKe]);
            matrices(i,(2*j+2),(2*j+3)) = hipCsub(matrices(i,(2*j+2),(2*j+3)), Ke[indexKe + 1]);
            matrices(i,(2*j+3),(2*j+3)) = hipCadd(matrices(i,(2*j+3),(2*j+3)), Ke[indexKe + 4]);

            for (int r=1; r<4; ++r){
                for (int c=0; c<r; ++c){
                    matrices(i,(2*j+r),(2*j+c)) = matrices(i,(2*j+c),(2*j+r));
                }
            }
        }

        kernel_Ke_halfspace_symm(alpha[n], beta[n], rho[n], c_test[i % velocities_length], k, Ke+indexKe);
        matrices(i, (2*n), (2*n))        = hipCadd(matrices(i, (2*n), (2*n)), Ke[indexKe]);
        matrices(i, (2*n), (2*n+1))      = hipCadd(matrices(i, (2*n), (2*n+1)), Ke[indexKe + 1]);
        matrices(i, (2*n+1), (2*n))      = matrices(i, (2*n), (2*n+1));
        matrices(i, (2*n+1), (2*n+1))    = hipCadd(matrices(i, (2*n+1), (2*n+1)), Ke[indexKe + 2]);
    }    
    
}

/* Modifies our test velocities so they are not too close to alpha or beta, causing numerical problems (MASWaves does this too).
    Parallel over one block.
*/
__global__ void kernel_too_close(int velocities_length, int nPlus, dfloat *c_test, dfloat *alpha, dfloat *beta, dfloat epsilon){

    int index = threadIdx.x;
    int blockSize = blockDim.x;

    for (int c=index; c<velocities_length; c+=blockSize){

        for (int i=0; i<nPlus; ++i){
            while (abs(c_test[c]-alpha[i]) < epsilon || abs(c_test[c]-beta[i]) < epsilon){
                c_test[c] *= 1-epsilon;
            }
        }
    }
}

/* Constructs a Ke Layer in a single CUDA thread, using CUDA complex numbers. Note Ke is length 6.
    This is because we have accounted for symmetry and other redundant entries in the Ke layer, reducing
    the memory requirements and number of computations.
*/
__device__ void kernel_Ke_layer_symm(dfloat r_h, dfloat r_alpha, dfloat r_beta, dfloat r_rho, dfloat r_c_test, dfloat r_k, hipDoubleComplex *Ke){

    // Idea: make rsquare and ssquare real, then take the square root of their absolute value.
    // Make them into complex numbers. If negative, then initialize them as imaginary, real if positive.
    // Use formulas for sinh,... of complex numbers

    hipDoubleComplex h       = make_hipDoubleComplex(r_h, 0.0);
    hipDoubleComplex alpha   = make_hipDoubleComplex(r_alpha, 0.0);
    hipDoubleComplex beta    = make_hipDoubleComplex(r_beta, 0.0);
    hipDoubleComplex rho     = make_hipDoubleComplex(r_rho, 0.0);
    hipDoubleComplex c_test  = make_hipDoubleComplex(r_c_test, 0.0);
    hipDoubleComplex k       = make_hipDoubleComplex(r_k, 0.0);

    hipDoubleComplex r, s;

    dfloat rSquare = 1.0 - (r_c_test*r_c_test)/(r_alpha*r_alpha);
    dfloat sSquare = 1.0 - (r_c_test*r_c_test)/(r_beta*r_beta);

    if (rSquare<0.0){
        r = make_hipDoubleComplex(0, sqrt(-1.0*rSquare));
    }
    else{
        r = make_hipDoubleComplex(sqrt(rSquare), 0);
    }
    if (sSquare<0.0){
        s = make_hipDoubleComplex(0, sqrt(-1.0*sSquare));
    }
    else{
        s = make_hipDoubleComplex(sqrt(sSquare), 0);
    }

    hipDoubleComplex rProduct = hipCmul(hipCmul(k,r),h);
    hipDoubleComplex sProduct = hipCmul(hipCmul(k,s),h);
    
    hipDoubleComplex Cr = make_hipDoubleComplex(cosh(hipCreal(rProduct))*cos(hipCimag(rProduct)), sinh(hipCreal(rProduct))*sin(hipCimag(rProduct)));
    hipDoubleComplex Sr = make_hipDoubleComplex(sinh(hipCreal(rProduct))*cos(hipCimag(rProduct)), cosh(hipCreal(rProduct))*sin(hipCimag(rProduct)));
    hipDoubleComplex Cs = make_hipDoubleComplex(cosh(hipCreal(sProduct))*cos(hipCimag(sProduct)), sinh(hipCreal(sProduct))*sin(hipCimag(sProduct)));
    hipDoubleComplex Ss = make_hipDoubleComplex(sinh(hipCreal(sProduct))*cos(hipCimag(sProduct)), cosh(hipCreal(sProduct))*sin(hipCimag(sProduct)));
    
    hipDoubleComplex One = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex neg = make_hipDoubleComplex(-1.0, 0.0);
    hipDoubleComplex Two = make_hipDoubleComplex(2.0, 0.0);

    hipDoubleComplex D = hipCadd(hipCmul(Two, hipCsub(One,hipCmul(Cr,Cs))), hipCmul(hipCmul(hipCadd(hipCdiv(One,hipCmul(r,s)), hipCmul(r,s)), Sr), Ss));

    // Now we make the 4x4 matrix as a 1D array and fill it in

    hipDoubleComplex krcd = hipCdiv(hipCmul(hipCmul(hipCmul(k,rho),c_test),c_test), D);

    Ke[0]   =   hipCmul(krcd, hipCsub(hipCdiv(hipCmul(Cr,Ss),s), hipCmul(r,hipCmul(Sr,Cs))));
    Ke[1]   =   hipCsub(hipCmul(krcd, hipCsub(hipCsub(hipCmul(Cr,Cs), hipCmul(hipCmul(r,s),hipCmul(Sr,Ss))), One)), hipCmul(k,hipCmul(rho,hipCmul(beta,hipCmul(beta,hipCadd(One,hipCmul(s,s)))))));
    Ke[2]   =   hipCmul(krcd, hipCsub(hipCmul(r,Sr), hipCdiv(Ss,s)));
    Ke[3]   =   hipCmul(krcd, hipCsub(Cs,Cr));
    
    Ke[4]   =   hipCmul(krcd, hipCsub(hipCdiv(hipCmul(Sr,Cs),r), hipCmul(hipCmul(s,Cr),Ss)));
    Ke[5]   =   hipCmul(krcd, hipCsub(hipCmul(s,Ss), hipCdiv(Sr,r)));
}

/* Creates the information for the final Ke halfspace, using cuda complex numbers.
    Note Ke is length 3 since it is symmetric in this case.
*/
__device__ void kernel_Ke_halfspace_symm(dfloat r_alpha, dfloat r_beta, dfloat r_rho, dfloat r_c_test, dfloat r_k, hipDoubleComplex *Ke){

    hipDoubleComplex alpha   = make_hipDoubleComplex(r_alpha, 0.0);
    hipDoubleComplex beta    = make_hipDoubleComplex(r_beta, 0.0);
    hipDoubleComplex rho     = make_hipDoubleComplex(r_rho, 0.0);
    hipDoubleComplex c_test  = make_hipDoubleComplex(r_c_test, 0.0);
    hipDoubleComplex k       = make_hipDoubleComplex(r_k, 0.0);

    dfloat rSquare = 1.0 - (r_c_test*r_c_test)/(r_alpha*r_alpha);
    dfloat sSquare = 1.0 - (r_c_test*r_c_test)/(r_beta*r_beta);

    hipDoubleComplex r, s;

    if (rSquare<0.0){
        r = make_hipDoubleComplex(0, sqrt(-1.0*rSquare));
    }
    else{
        r = make_hipDoubleComplex(sqrt(rSquare), 0);
    }
    if (sSquare<0.0){
        s = make_hipDoubleComplex(0, sqrt(-1.0*sSquare));
    }
    else{
        s = make_hipDoubleComplex(sqrt(sSquare), 0);
    }

    hipDoubleComplex One = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex Two = make_hipDoubleComplex(2.0, 0.0);
    hipDoubleComplex temp = hipCdiv(hipCmul(hipCmul(hipCmul(k,rho),hipCmul(beta,beta)),hipCsub(One,hipCmul(s,s))), hipCsub(One,hipCmul(r,s)));

    Ke[0] = hipCmul(temp, r);
    Ke[1] = hipCsub(temp, hipCmul(Two, hipCmul(hipCmul(k,rho),hipCmul(beta,beta))));
    Ke[2] = hipCmul(temp, s);
}

