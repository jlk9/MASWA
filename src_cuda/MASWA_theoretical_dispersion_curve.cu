#include "hip/hip_runtime.h"
#include "MASW.cuh"

// Written by Joseph Kump (josek97@vt.edu). Last modified 01/27/2020

/* Implements the theoretical dispersion curve in CUDA */

/* Carries out the work of MASW inversion. Gets the stiffness matrix determinants, and
    finds the first sign change in each wavelength's array of determinants.
 
    Inputs:
    curve       the dispersion curve struct
    
    Outputs:
        void, but sets the theoretical dispersion curve values in the curve_t object (c_t
            and lambda_t), and prints the misfit between the theoretical and experimental
            velocities.
 */
void MASWA_theoretical_dispersion_curve_CUDA(curve_t *curve){
    
    
    // Initialize stiffness matrices on kernel here. They're first allocated as a 1D array,
    // then pointers are assigned to each matrix:
    hipDoubleComplex **d_matrices;

    int size = 2*(curve->n+1);
    hipDoubleComplex     *d_data;
    hipMalloc((void**)&d_matrices, curve->curve_length*curve->velocities_length*sizeof(hipDoubleComplex*));
    hipMalloc(&d_data, size*size*curve->curve_length*curve->velocities_length*sizeof(hipDoubleComplex));
    kernel_assign_matrices_to_data<<<1,1>>>(d_matrices, d_data, curve->curve_length, curve->velocities_length, curve->n);

    // Fill in the stiffness matrix entries, then use Gaussian elimination to make finding
    // determinants easy:
    MASWA_stiffness_matrix_CUDA(curve, d_matrices);

    // The determinants for each wavelength are stored in the same block. Since the number
    // of test velocities is usually greater than the block size, multiple blocks will
    // usually be assigned to each wavelength.
    const int blockSize     = 256;
    int blocksPerWavelength = (curve->velocities_length / blockSize)+1;

    // Neighbors holds the determinant of the "next block" for the wavelength, to make
    // sure sign changes between blocks are not ignored. signChange holds the first
    // determinant sign change for each block.
    int *d_neighbors, *d_signChange;
    hipMalloc(&d_neighbors, blocksPerWavelength*curve->curve_length*sizeof(int));
    hipMalloc(&d_signChange, blocksPerWavelength*curve->curve_length*sizeof(int));

    // Breaks up the blocks along two axes, assigning blocks in order to each wavelength:
    dim3 numBlocks(curve->curve_length, blocksPerWavelength);

    // Find the first determinant sign change for each wavelength within its block only:
    kernel_block_sign_change<<<numBlocks, 256, 256>>>(d_neighbors, d_signChange, curve->velocities_length, size, d_matrices);

    dfloat *d_c_t, *d_c_test;
    hipMalloc(&d_c_t, curve->curve_length*sizeof(dfloat));
    hipMalloc(&d_c_test, curve->velocities_length*sizeof(dfloat));
    hipMemcpy(d_c_test, curve->c_test, curve->velocities_length*sizeof(dfloat), hipMemcpyHostToDevice);
    
    // Finds the first sign change for each wavelength across all blocks, then assigns
    // the corresponding test velocity to that entry of the theoretical dispersion curve:
    kernel_first_sign_change<<<1, 1>>>(d_c_t, d_c_test, d_signChange, blocksPerWavelength, curve->curve_length);
    hipMemcpy(curve->c_t, d_c_t, curve->curve_length*sizeof(dfloat), hipMemcpyDeviceToHost);

    // Compute the misfit. It's quicker to do it here where the dispersion curves are
    // allocated to the GPU anyway:
    int misfitBlocks = (curve->curve_length + blockSize - 1) / blockSize;

    dfloat *error, *d_error, *d_c_curve0;
    error = (dfloat*) calloc(1, sizeof(dfloat));
    hipMalloc(&d_error, misfitBlocks*sizeof(dfloat));
    hipMalloc(&d_c_curve0, curve->curve_length*sizeof(dfloat));
    hipMemcpy(d_c_curve0, curve->c_curve0, curve->curve_length*sizeof(dfloat), hipMemcpyHostToDevice);

    kernel_misfit_00<<<1, blockSize, blockSize>>>(curve->curve_length, d_c_t, d_c_curve0, d_error);

    hipMemcpy(error, d_error, sizeof(dfloat), hipMemcpyDeviceToHost);

    error[0] = (error[0]*100.0) / (dfloat) curve->curve_length;

    printf("error is %f\n", error[0]);

    hipFree(d_data);
    hipFree(d_matrices);
    hipFree(d_neighbors);
    hipFree(d_signChange);
    hipFree(d_c_t);
    hipFree(d_c_test);
    hipFree(d_error);
    hipFree(d_c_curve0);
    
}

/* Identifies the first dispersion curve with a sign change in each block of velocity values.

    Inputs:
    neighbors           stores the determinant for the first entry of the "next block over"
                            to make sure sign change between blocks aren't ignored
    signChange          stores the index of the fist sign change in each block, filled by
                            this kernel
    velocities_length   the length of the test velocities array
    size                the axis size of the stiffness matrices (2*(n+1))
    matrices            the 2D array that holds the stiffness matrix entries
    
    Output:
    void, stores the index of the first determinant sign change for each block of each
        wavelength in signChange
*/
__global__ void kernel_block_sign_change(int *neighbors, int *signChange, int velocities_length, int size, hipDoubleComplex **matrices){
    /*
    IDEA: break up threads/blocks by c_test and lambda_curve0

    Each thread fills in one stiffness matrix corresponding to its lambda_curve0 and c_test values (thus we can parallelize ke_layer and ke_halfspace to fill these in)
    */

    // Both signChange and neighbors are size curve_length * blocksPerWavelength

    static const int blockSize  = 256;
    int threadIndex             = threadIdx.x;
    // We get the thread's matching wavelength value, which is its block id in the x axis:
    int wavelengthIndex         = blockIdx.x;
    // Then we get its velocity value, which is determined by its thread id and block id in the y axis:
    int blockIndexY             = blockIdx.y;
    int velocityIndex           = blockIndexY * blockSize + threadIndex;
    int blocksPerWavelength     = gridDim.y;

    // The portion of determinants stored in this block.
    __shared__ dfloat e[blockSize];

    e[threadIndex] = 0;
    if (velocityIndex < velocities_length){
        
        //This is where we get the determinants from the reduced matrices:
        hipDoubleComplex det = matrices[wavelengthIndex*velocities_length + velocityIndex][0];
        for (int i=1; i<size; ++i){
            // We only need to multiply the diagonal entries:
            det = hipCmul(matrices[wavelengthIndex*velocities_length + velocityIndex][i*size + i], det);
        }

        e[threadIndex] = hipCreal(det);
    }

    __syncthreads();

    #define neighbors(r, c) (neighbors[(r)*blocksPerWavelength + (c)])
    #define signChange(r, c) (signChange[(r)*blocksPerWavelength + (c)])
    if (threadIndex == 0){
        // Need the next nearest neighbor for sign change comparison, so the first entry of each block is stored in global memory.
        neighbors(wavelengthIndex, blockIndexY) = e[threadIndex];
        signChange(wavelengthIndex, blockIndexY) = -1;

        for (int i=0; i<blockSize-1; ++i){
            // If we find a sign change, we set the first sign change in this range of test velocities to that index and break:
            if (e[i]*e[i+1] < 0){
                signChange(wavelengthIndex, blockIndexY) = blockIndexY*blockSize + i + 1;
                break;
            }
        }

        
        // If we don't, we then check the first entry in the next range of test velocities for a sign change:
        if (signChange(wavelengthIndex, blockIndexY) == -1 && blockIndexY != blocksPerWavelength-1
            && neighbors(wavelengthIndex, blockIndexY+1)*e[blockSize-1] < 0){

            signChange(wavelengthIndex, blockIndexY) = blockIndexY*(blockSize+1);
        }
        
    }

}

/* Searches through array to find the first sign change for each wavelength.
    This iterates over multiple blocks from kernel_block_sign_change.
    Currently a serial kernel, but it has a negligible effect on the runtime. May be
    parallelized later.
    
    Inputs:
    c_t                     the theoretical dispersion curve
    c_test                  the test velocity array
    signChange              the indices of the first sign changes for each block of each
                                wavelength
    blocksPerWavelength     the number of blocks assigned to each wavelength
    curve_length            the length of the dispersion curve
    
    Output:
    void, but assigns the right theoretical velocity to each wavelength based on the
        first overall sign change
*/
__global__ void kernel_first_sign_change(dfloat *c_t, dfloat* c_test, int *signChange, int blocksPerWavelength, int curve_length){

    #define signChange(r, c) (signChange[(r)*blocksPerWavelength + (c)])
    
    // For each entry in the dispersion curve-
    for (int i=0; i<curve_length; ++i){
        // -we iterate over all test velocities (in their blocks)-
        for (int j=0; j<blocksPerWavelength; ++j){
            if (signChange(i,j) != -1){
                // -and stop when we find the first determinant with a sign change from
                // its predecessor.
                c_t[i] =  c_test[signChange(i,j)];
                break;
            }
        }
    }

}





