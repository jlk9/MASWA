#include "hip/hip_runtime.h"
#include "MASW.cuh"

/* Implements the misfit function in CUDA */

/* Runs the kernel for computing the misfit. Not used for the inversion (since it's more efficient to
    just compute the misfit with the dispersion curves while they're already on the GPU), but still
    useful for testing purposes. */
dfloat run_misfit(int curve_length, dfloat *c_t, dfloat *c_curve0){

    dfloat *d_ct, *d_ccurve0, *d_error0, *d_error1, *error;

    error = (dfloat*) calloc(1, sizeof(dfloat));
    hipMalloc(&d_ct, curve_length*sizeof(dfloat));
    hipMalloc(&d_ccurve0, curve_length*sizeof(dfloat));
    hipMalloc(&d_error0, sizeof(dfloat));

    hipMemcpy(d_ct, c_t, curve_length*sizeof(dfloat), hipMemcpyHostToDevice);
    hipMemcpy(d_ccurve0, c_curve0, curve_length*sizeof(dfloat), hipMemcpyHostToDevice);

    int blockSize = 256;

    // First testing the speed of the single-block implementation:
    double time_spent0 = 0.0;
    clock_t begin0 = clock();

    kernel_misfit_00<<<1, blockSize, blockSize>>>(curve_length, d_ct, d_ccurve0, d_error0);

    clock_t end0 = clock();
	time_spent0 += (double)(end0 - begin0) / CLOCKS_PER_SEC;
    printf("Time for misfit with one block is %f\n", time_spent0);

    hipMemcpy(error, d_error0, sizeof(dfloat), hipMemcpyDeviceToHost);

    // Now testing the speed of the multi-block implementation:
    int numBlocks = (curve_length + blockSize - 1) / blockSize;
    hipMalloc(&d_error1, numBlocks*sizeof(dfloat));

    double time_spent1 = 0.0;
    clock_t begin1 = clock();

    kernel_misfit_01<<<numBlocks, blockSize, blockSize>>>(curve_length, d_ct, d_ccurve0, d_error1);
    kernel_misfit_block_summation<<<1, blockSize, blockSize>>>(numBlocks, d_error1);

    clock_t end1 = clock();
	time_spent1 += (double)(end1 - begin1) / CLOCKS_PER_SEC;
    printf("Time for misfit with multiple blocks is %f\n", time_spent1);

    hipMemcpy(error, &d_error1[0], sizeof(dfloat), hipMemcpyDeviceToHost);

    hipFree(d_ct);
    hipFree(d_ccurve0);
    hipFree(d_error0);
    hipFree(d_error1);

    return error[0];
}

/* Global kernel, computes the misfit between the real and theoretical curves. Since the dispersion curve
    is typically small, it is usually more efficient to compute the misfit within a single block. */
__global__ void kernel_misfit_00(const int curve_length, dfloat *c_t, dfloat *c_curve0, dfloat *error){

    static const int blockSize = 256; //We're treating blocksize as 256 for now

    int index = threadIdx.x;
    int stride = blockSize;

    //error[0] = 0.0;

    //extern __shared__ dfloat e[];
    __shared__ dfloat e[blockSize];

    e[index] = 0.0;

    for (int i=index; i<curve_length; i+=stride){
        e[index] += sqrt((c_curve0[i]-c_t[i])*(c_curve0[i]-c_t[i])) / c_curve0[i];
    }

    __syncthreads();
    for (int size = stride/2; size>0; size/=2) { //uniform
        if (index<size)
            e[index] += e[index+size];
        __syncthreads();
    }
    error[0] = e[0];

}

/* Global kernel, computes the misfit between the real and theoretical curves over multiple blocks. Need
    to use kernel_misfit_block_summation below to sum up these block misfits. Generally the single block
    implementation is more efficient, so that is used by default. */
__global__ void kernel_misfit_01(const int curve_length, dfloat *c_t, dfloat *c_curve0, dfloat *error){

    static const int blockSize = 256; //We're treating blocksize as 256 for now

    int threadIndex = threadIdx.x;
    int gridIndex = threadIndex + blockIdx.x*blockSize;
    int stride = blockSize * gridDim.x;

    //error[0] = 0.0;

    //extern __shared__ dfloat e[];
    __shared__ dfloat e[blockSize];

    e[threadIndex] = 0.0;

    for (int i=gridIndex; i<curve_length; i+=stride){
        e[threadIndex] += sqrt((c_curve0[i]-c_t[i])*(c_curve0[i]-c_t[i])) / c_curve0[i];
    }

    // Reduce within the blocks:
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (threadIndex<size)
            e[threadIndex] += e[threadIndex+size];
        __syncthreads();
    }

    if (threadIndex == 0){
        error[blockIdx.x] = e[0];
    }
}

/* Sums all the entries in an array and stores them in the first entry. This is used to add up
    multiple blocks' worth of memory for the multi-block misfit. */
__global__ void kernel_misfit_block_summation(const int array_length, dfloat *array){

    static const int blockSize = 256;
    int index = threadIdx.x;
    int stride = blockSize;

    __shared__ dfloat e[blockSize];
    e[index] = 0.0;

    for (int i=index; i<array_length; i+=stride){
        e[index] += array[i];
    }

    __syncthreads();
    for (int size = stride/2; size>0; size/=2) { //uniform
        if (index<size)
            e[index] += e[index+size];
        __syncthreads();
    }
    array[0] = e[0];

}

